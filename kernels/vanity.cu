#include "hip/hip_runtime.h"
#include <stdio.h>
#include "base58.h"
#include "vanity.h"
#include "sha256.h"

// ------------------------------------------------------------------
// XorShift128+ PRNG state & helper functions (fast per-thread RNG)
struct xorshift128plus_state {
    uint64_t s[2];
};

__device__ void init_xorshift(xorshift128plus_state &st,
                              const uint8_t *seed,   // 32-byte GPU seed
                              uint64_t idx)
{
    // Extract all four 64-bit values from the 32-byte seed
    uint64_t k0 = *((const uint64_t*)(seed + 0));
    uint64_t k1 = *((const uint64_t*)(seed + 8));
    uint64_t k2 = *((const uint64_t*)(seed + 16));
    uint64_t k3 = *((const uint64_t*)(seed + 24));

    // Mix k0 and k2 with idx for s[0]
    uint64_t z0 = k0 ^ k2;  // Combine both parts
    z0 += idx;
    z0 = (z0 ^ (z0 >> 30)) * 0xbf58476d1ce4e5b9ULL;
    z0 = (z0 ^ (z0 >> 27)) * 0x94d049bb133111ebULL;
    st.s[0] = z0 ^ (z0 >> 31);

    // Mix k1 and k3 with idx (and golden ratio) for s[1]
    uint64_t z1 = k1 ^ k3;  // Combine both parts
    z1 += idx + 0x9e3779b97f4a7c15ULL;
    z1 = (z1 ^ (z1 >> 30)) * 0xbf58476d1ce4e5b9ULL;
    z1 = (z1 ^ (z1 >> 27)) * 0x94d049bb133111ebULL;
    st.s[1] = z1 ^ (z1 >> 31);
}

__device__ uint64_t xorshift128plus_next(xorshift128plus_state &st) {
    uint64_t s1 = st.s[0], s0 = st.s[1];
    uint64_t result = s0 + s1;
    st.s[0] = s0;
    s1 ^= s1 << 23;
    st.s[1] = (s1 ^ s0 ^ (s1 >> 18) ^ (s0 >> 5));
    return result;
}

__device__ int done = 0;
__device__ unsigned long long count = 0;
__device__ bool d_case_insensitive = false;

// Simplified ed25519 key generation for GPU
__device__ void generate_ed25519_keypair(const uint8_t *seed32, uint8_t *private_key, uint8_t *public_key) {
    // Use the seed directly as private key (simplified approach)
    memcpy(private_key, seed32, 32);
    
    // Generate deterministic public key from private key using SHA256
    // This is a simplified version - real ed25519 uses curve25519 point multiplication
    CUDA_SHA256_CTX ctx;
    cuda_sha256_init(&ctx);
    cuda_sha256_update(&ctx, private_key, 32);
    // Add some entropy to make it more random
    cuda_sha256_update(&ctx, seed32, 32);
    cuda_sha256_final(&ctx, public_key);
}

__global__ void vanity_keypair_search(uint8_t *buffer, uint64_t stride) {
    // Deconstruct buffer - FIX: Use proper casting for memory access
    uint8_t *seed = buffer;
    
    // FIX: Proper memory access for target_len
    uint64_t target_len = *((uint64_t*)(buffer + 32));
    char *target = (char*)(buffer + 40);
    
    // FIX: Proper memory access for suffix_len  
    uint64_t suffix_len = *((uint64_t*)(buffer + 40 + target_len));
    char *suffix = (char*)(buffer + 40 + target_len + 8);
    
    uint8_t *out = buffer + 40 + target_len + suffix_len + 8;

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize XorShift128+ state
    xorshift128plus_state st;
    init_xorshift(st, seed, idx);

    for (uint64_t iter = 0; iter < uint64_t(1000) * 1000; iter++) {
        // Has someone found a result?
        if (iter % 100 == 0) {
            if (atomicMax(&done, 0) == 1) {
                atomicAdd(&count, iter);
                return;
            }
        }

        // Generate a 32-byte seed for ed25519 keypair
        uint8_t keypair_seed[32];
        for (int i = 0; i < 4; ++i) {
            uint64_t rnd = xorshift128plus_next(st);
            memcpy(&keypair_seed[i * 8], &rnd, 8);
        }

        // Generate ed25519 keypair
        uint8_t private_key[32];
        uint8_t public_key[32];
        generate_ed25519_keypair(keypair_seed, private_key, public_key);

        // Encode public key to base58
        unsigned char encoded_pubkey[44] = {0};
        ulong encoded_len = fd_base58_encode_32(public_key, encoded_pubkey, d_case_insensitive);

        // Check if it matches our target
        if (matches_target(encoded_pubkey, (unsigned char*)target, target_len, (unsigned char*)suffix, suffix_len, encoded_len)) {
            // Are we first to write result?
            if (atomicMax(&done, 1) == 0) {
                // Copy private key and public key to output
                memcpy(out, private_key, 32);      // First 32 bytes: private key
                memcpy(out + 32, public_key, 32);  // Next 32 bytes: public key
                atomicAdd(&count, iter + 1);       // Only increment count on actual match
            }
            return;
        }
    }
}

__device__ bool matches_target(unsigned char *a, unsigned char *target, uint64_t n, unsigned char *suffix, uint64_t suffix_len, ulong encoded_len)
{
    for (int i = 0; i < n; i++)
    {
        if (a[i] != target[i])
            return false;
    }
    for (int i = 0; i < suffix_len; i++)
    {
        if (a[encoded_len - suffix_len + i] != suffix[i])
            return false;
    }
    return true;
}

// Simple GPU globals for result storage
__device__ bool gpu_found = false;
__device__ uint8_t gpu_result[64]; // 32 bytes private + 32 bytes public
__device__ uint64_t gpu_iterations = 0;

__global__ void gpu_vanity_search(const uint8_t* seed, const char* target, uint64_t target_len, uint64_t max_iterations) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if result already found
    if (gpu_found) return;
    
    // Initialize random state per thread
    xorshift128plus_state rng_state;
    init_xorshift(rng_state, seed, idx);
    
    for (uint64_t iter = 0; iter < max_iterations && !gpu_found; iter++) {
        // Generate random 32-byte private key
        uint8_t private_key[32];
        for (int i = 0; i < 4; i++) {
            uint64_t rand_val = xorshift128plus_next(rng_state);
            memcpy(&private_key[i * 8], &rand_val, 8);
        }
        
        // Generate public key (simplified - just hash private key)
        uint8_t public_key[32];
        CUDA_SHA256_CTX ctx;
        cuda_sha256_init(&ctx);
        cuda_sha256_update(&ctx, private_key, 32);
        cuda_sha256_final(&ctx, public_key);
        
        // Convert to base58
        unsigned char base58_pubkey[64];
        ulong base58_len = fd_base58_encode_32(public_key, base58_pubkey, false);
        
        // Check if matches target prefix
        bool matches = true;
        for (uint64_t i = 0; i < target_len && i < base58_len; i++) {
            if (base58_pubkey[i] != (unsigned char)target[i]) {
                matches = false;
                break;
            }
        }
        
        if (matches) {
            // Atomic check-and-set to ensure only first thread writes result
            bool was_found = atomicExch(&gpu_found, true);
            if (!was_found) {
                // Copy result to global memory
                memcpy(gpu_result, private_key, 32);
                memcpy(gpu_result + 32, public_key, 32);
                atomicAdd((unsigned long long*)&gpu_iterations, iter + 1);
            }
            return;
        }
    }
    
    // Add iterations even if no match found
    atomicAdd((unsigned long long*)&gpu_iterations, max_iterations);
}

extern "C" void vanity_keypair_round(
    int gpu_id,
    uint8_t *seed,
    char *target,
    char *suffix,
    uint64_t target_len,
    uint64_t suffix_len,
    uint8_t *out,
    bool case_insensitive)
{
    // Set device
    hipSetDevice(gpu_id);
    
    // Reset GPU state
    bool found_init = false;
    uint64_t iter_init = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_found), &found_init, sizeof(bool));
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_iterations), &iter_init, sizeof(uint64_t));
    
    // Launch kernel with reasonable block/thread counts
    int threads_per_block = 256;
    int num_blocks = 1024;
    uint64_t max_iter_per_thread = 1000000; // 1M iterations per thread
    
    gpu_vanity_search<<<num_blocks, threads_per_block>>>(seed, target, target_len, max_iter_per_thread);
    
    // Wait for completion
    hipDeviceSynchronize();
    
    // Check if we found a result
    bool found;
    uint64_t total_iterations;
    hipMemcpyFromSymbol(&found, HIP_SYMBOL(gpu_found), sizeof(bool));
    hipMemcpyFromSymbol(&total_iterations, HIP_SYMBOL(gpu_iterations), sizeof(uint64_t));
    
    if (found) {
        // Copy result back to host
        hipMemcpyFromSymbol(out, HIP_SYMBOL(gpu_result), 64);
        // Store iteration count in last 8 bytes
        memcpy(out + 56, &total_iterations, 8);
    } else {
        // No result found, zero out the output
        memset(out, 0, 64);
    }
}